#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cassert>
#include <vector>
#include <ranges>
#include <limits>
#include "cmdline.h"
#include <sys/ioctl.h>
#include <unistd.h>
#include <hip/hip_complex.h>

typedef std::complex<float> complex;

// window in complex plane from z1 to z2 with ascii chars as display
class ascii_view {
  private:
    size_t cols_, rows_;
    complex z1_, z2_;
    // each point has an associated char ('*' for non-escaping, '0'-'9' for escaping)
    std::vector<char> data_;
  public:
    void set_data(const int* h_iter) {
        for (size_t i = 0; i < win.cols_ * win.rows_; ++i) {
            data_[i] = (h_iter[i] == 10 ? char(42) : char(h_iter[i]+48));
        }
    }
    void print() const {
        for (int n = 0; n < rows_; ++n) {
            for (char c : std::views::counted(data_.begin()+n*cols_, cols_)) {
                std::cout << c;
            }
            std::cout << std::endl;
        }
    }
    // construct complex plane window z1 to z2
    ascii_view(size_t cols, size_t rows, complex z1, complex z2)
        : cols_{cols}, rows_{rows}, z1_{z1}, z2_{z2} {
            assert(cols_ > 1);
            assert(rows_ > 1);
            data_.resize(cols_*rows_);
            for(char& c : data_) {
                c = '.';
            }
    }
};

// set win.dat_ member from h_d

void usage(const std::string& name) {
    std::cout << "Usage: " << std::endl << "  " << name << " cols rows+1 [-noprint]" << std::endl;
    std::cout << "     where: cols >= 2, rows+1 >=3" << std::endl;
    std::cout << "I.e: " << std::endl << "  " << name << " $(tput cols) $(tput lines)" << std::endl;
}

// kernel to iterate one point in the c-plane, returns iterations to escape
__global__ void point_iterate(int* area, float zr1, float zc1, float zr2, float zc2, const int nx, const int ny, const int max_iter) {
    int i = 0;
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    //printf(" %d %d \n", ix, iy);
    if (ix >= nx || iy >= ny) {
        return;
    }
    float zr = (zr2 - zr1) * ix / (nx - 1.0);
    float zc = (zc2 - zc1) * iy / (ny - 1.0));
    float cr = z1r + zr; 
    float cc = z1c + zc;

    //printf(" %e %e \n", hipCrealf(c), hipCimagf(c));
    while ( (zr*zr + zc*zc) < 4.0 && i < max_iter) {
        zr = zr*zr - zc*zc + cr;
        zc = 2.0*zr*zc + cc;
        i++;
    }
    printf(" %d \n", i);
    area[idx] = i;
};    

int main(int argc, char *argv[]) {
    // get terminal size
    struct winsize w;
    ioctl(STDOUT_FILENO, TIOCGWINSZ, &w);

    cmdline::parser a;
    a.add<int>("cols", 'c', "number of columns", false, w.ws_col, cmdline::range(2, std::numeric_limits<int>::max()));
    a.add<int>("rows", 'r', "number of rows", false, w.ws_row-1, cmdline::range(2, 1000000));
    a.add<int>("bsize", 'b', "block size", false, 8, cmdline::range(1, 32));
    a.add("noprint", '\0', "supress printing");
    a.parse_check(argc, argv);

    size_t cols = a.get<int>("cols");
    size_t rows = a.get<int>("rows");
    size_t bsize = a.get<int>("bsize");
    bool noprint = a.exist("noprint");

    std::cout << "Calculating mandelbrot set with size "  << cols 
        << " * " << rows  << std::endl;

    // set up 2D block sizes
    dim3 block(bsize, bsize);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    std::cout << "block: " << block.x << " " << block.y << std::endl;
    std::cout << "grid: " << grid.x << " " << grid.y << std::endl;

    // view window for mandelbrot set
    complex z1{-2.0,-1.0};
    complex z2{1.0,1.0};

    // malloc host memory for result of mandelbrot set and initialize to 0
    int* h_m = (int *)malloc(sizeof(int)*rows*cols);
    memset(h_m, 0, sizeof(int)*rows*cols);

    int* d_m;
    // malloc device memory for result of mandelbrot set (number of iterations for each point)
    hipMalloc((void **) &d_m, sizeof(int)*rows*cols);

    // call kernel
    point_iterate<<< grid, block >>>(d_m, z1.real(), z1.imag(), z2.real(), z2.imag(), cols, rows, 100);
    hipDeviceSynchronize();

    // copy result from device to host and free device memory
    hipMemcpy(h_m, d_m, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
    hipFree(d_m);

    // print result
    if (!noprint) {
        ascii_view win(cols,rows,z1,z2);
        win.set_data(h_m);
        win.print();  
    }
    // free host memory
    free(h_m);
    return 0;
}
