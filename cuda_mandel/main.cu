#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cassert>
#include <vector>
#include <ranges>
#include <limits>
#include "cmdline.h"
#include <sys/ioctl.h>
#include <unistd.h>
#include <hip/hip_complex.h>

//#include <stdio.h>
//#include <unistd.h>

typedef std::complex<float> complex;

// window in complex plane from z1 to z2 with ascii chars as display
class ascii_view {
  private:
    size_t cols_, rows_;
    complex z1_, z2_;
    // each point has an associated char ('*' for non-escaping, '0'-'9' for escaping)
    std::vector<char> data_;
    // return complex number in window for given row and column
    complex get_z(int row_index, int col_index) const {
        std::complex z = z2_-z1_;
        z.real(z.real() * col_index / (cols_ - 1.0));
        z.imag(z.imag() * row_index / (rows_ - 1.0));
        return z1_ + z;
    }
    // return complex number in window for given linear index
    complex get_z(int linear_index) const {
        int row_index = linear_index / cols_;
        int col_index = linear_index % cols_;
        return get_z(row_index, col_index);
    }
    // external calculator function
    friend void calculate_tile(size_t tile_size, size_t offset, ascii_view& win, auto fun);
  public:
    void print() const {
        for (int n = 0; n < rows_; ++n) {
            for (char c : std::views::counted(data_.begin()+n*cols_, cols_)) {
                std::cout << c;
            }
            std::cout << std::endl;
        }
    }
    // construct complex plane window z1 to z2
    ascii_view(size_t cols, size_t rows, complex z1, complex z2)
        : cols_{cols}, rows_{rows}, z1_{z1}, z2_{z2} {
            assert(cols_ > 1);
            assert(rows_ > 1);
            data_.resize(cols_*rows_);
            for(char& c : data_) {
                c = '.';
            }
    }
};

// calculates tile n of all tiles with size tile_size of the mandelbrot set
void calculate_tile(size_t tile_size, size_t offset, ascii_view& win, auto fun) {
    //std::cout << "calculate_tile: " << tile_size << " " << offset << std::endl;
    for (size_t i = 0; i < tile_size; ++i) {
        auto c = win.get_z(offset + i);
        auto iter = fun(c, 100)/10;
        win.data_[offset + i] = (iter == 10 ? char(42) : char(iter+48));
    }
}

void usage(const std::string& name) {
    std::cout << "Usage: " << std::endl << "  " << name << " cols rows+1 tiles [-noprint]" << std::endl;
    std::cout << "     where: cols >= 2, rows+1 >=3, ntiles >= 1" << std::endl;
    std::cout << "I.e: " << std::endl << "  " << name << " $(tput cols) $(tput lines) 1" << std::endl;
}

// lambda to iterate one point in the c-plane, returns iterations to escape
__global__ void point_iterate(int* area, float zr1, float zc1, float zr2, float zc2, const int nx, const int ny, const int max_iter) {
    int i = 0;
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    //printf(" %d %d \n", ix, iy);
    hipFloatComplex z1 = make_hipFloatComplex(zr1, zc1);
    hipFloatComplex z2 = make_hipFloatComplex(zr2, zc2);
    hipFloatComplex z = hipCsubf(z2,z1);
    z = make_hipFloatComplex(hipCrealf(z) * ix / (nx - 1.0), hipCimagf(z) * iy / (ny - 1.0));
    hipFloatComplex c = hipCaddf(z1, z);

    //printf(" %e %e \n", hipCrealf(c), hipCimagf(c));
    while (hipCabsf(z) < 2.0 && i < max_iter) {
        z = hipCaddf(hipCmulf(z,z), c);
        i++;
    }
    printf(" %d \n", i);
    area[idx] = i;
};    

int main(int argc, char *argv[]) {
    // get terminal size
    struct winsize w;
    ioctl(STDOUT_FILENO, TIOCGWINSZ, &w);

    cmdline::parser a;
    a.add<int>("cols", 'c', "number of columns", false, w.ws_col, cmdline::range(2, std::numeric_limits<int>::max()));
    a.add<int>("rows", 'r', "number of rows", false, w.ws_row-1, cmdline::range(2, 1000000));
    a.add("noprint", '\0', "supress printing");
    a.parse_check(argc, argv);

    size_t cols = a.get<int>("cols");
    size_t rows = a.get<int>("rows");
    bool noprint = a.exist("noprint");

    std::cout << "Calculating mandelbrot set with size "  << cols 
        << " * " << rows  << std::endl;

    // set up 2D block sizes
    dim3 block(32, 32);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    // view window for mandelbrot set
    complex z1{-2.0,-1.0};
    complex z2{1.0,1.0};
    // initialize ascii view
    ascii_view win(cols,rows,z1,z2);

    // malloc host memory for result of mandelbrot set
    int* h_m = (int *)malloc(sizeof(int)*rows*cols);
    // initialize memory to 0
    memset(h_m, 0, sizeof(int)*rows*cols);

    int* d_m;
    // malloc device memory for result of mandelbrot set
    hipMalloc((void **) &d_m, sizeof(int)*rows*cols);

    point_iterate<<< grid, block >>>(d_m, z1.real(), z1.imag(), z2.real(), z2.imag(), cols, rows, 100);
    hipDeviceSynchronize();

    // copy result from device to host
    hipMemcpy(h_m, d_m, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
    // free device memory
    hipFree(d_m);
 
    // print result
    if (!noprint) {
        win.print();  
    }
    // free host memory
    free(h_m);
    return 0;
}
